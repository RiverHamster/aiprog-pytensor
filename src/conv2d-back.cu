#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "macros.h"
#include "tensor.h"
#include <cassert>
#include <cstdio>

namespace ten {
// NB: we interchanged the meaning of C and K, and C now refers to the
// dimension of the input of backward pass (y), to maintain code consistency
template <ssize_t CBLK, ssize_t HBLK, ssize_t WBLK, ssize_t KBLK>
KERNEL conv2d_3x3_dx_ker(const float *y, const float *ker, float *dx, int C,
                         int nblkC, int H, int nblkH, int W, int nblkW, int K,
                         int nblkK) {
    __shared__ float t_in[CBLK][HBLK + 2][WBLK + 2], t_ker[CBLK * 9][KBLK],
        col[HBLK * WBLK][CBLK * 9];

    int batch = blockIdx.x;
    int off_k = (batch % nblkK) * KBLK;
    batch /= nblkK;
    int off_c = (batch % nblkC) * CBLK;
    batch /= nblkC;
    int off_h = blockIdx.y * HBLK, off_w = blockIdx.z * WBLK;
    int tid = threadIdx.x;
    y += (ssize_t)batch * C * H * W;
    dx += (ssize_t)batch * K * H * W;

    // load data
    // 256 threads
    if (tid < CBLK * (HBLK + 2) * (WBLK + 2)) {
        int c = tid / ((HBLK + 2) * (WBLK + 2));
        int h = (tid / (WBLK + 2)) % (HBLK + 2);
        int w = tid % (WBLK + 2);
        // BACK PASS: shift the convolution window
        if (off_c + c < C && off_h + h - 2 < H && off_w + w - 2 < W &&
            off_h + h - 2 >= 0 && off_w + w - 2 >= 0)
            t_in[c][h][w] =
                y[(off_c + c) * H * W + (off_h + h - 2) * W + off_w + w - 2];
        else
            t_in[c][h][w] = 0.0;
    }

    // load kernel
    // 288 threads
    if (tid < KBLK * 9) {
        int phase = tid / KBLK;
        int k = tid % KBLK;
        // BACK PASS: reverse the kernel, and exchange C, K
        // kernel is (9, K, C)
        // TODO: irregular access patterns
        for (int c = 0; c < CBLK; ++c) {
            if (off_c + c < C && off_k + k < K)
                t_ker[c * 9 + (8 - phase)][k] =
                    ker[phase * K * C + (off_k + k) * C + off_c + c];
            else
                t_ker[c * 9 + (8 - phase)][k] = 0.0;
        }
    }
    __syncthreads();

    // im2col transformation
    // 324 threads
    if (tid < HBLK * WBLK * 9) {
        int phase = tid % 9, phase_h = phase / 3, phase_w = phase % 3;
        int _quot = tid / 9;
        int h = _quot / WBLK, w = _quot % WBLK;
        for (int c = 0; c < CBLK; ++c) {
            col[_quot][c * 9 + phase] = t_in[c][h + phase_h][w + phase_w];
        }
    }

    __syncthreads();

    // matrix multiplication and write-back
    // 288 threads
    if (tid < HBLK * WBLK * (KBLK / 4)) {
        int k0 = tid / (HBLK * WBLK), pos = tid % (HBLK * WBLK);
        int h = pos / WBLK, w = pos % WBLK;
        for (int kstep = 0; kstep < 4; ++kstep) {
            int k = kstep * (KBLK / 4) + k0;
            float sum = 0;
            for (int m = 0; m < 9 * CBLK; ++m) {
                sum += col[pos][m] * t_ker[m][k];
            }
            // TODO: use atomicAdd, optimize to reduction when necessary
            if (off_k + k < K && off_h + h < H && off_w + w < W)
                atomicAdd(
                    &dx[(off_k + k) * H * W + (off_h + h) * W + off_w + w],
                    sum);
        }
    }
}

Tensor conv2d_3x3_grad_x(const Tensor &dy, const Tensor &ker) {
    constexpr int CBLK = 4, HBLK = 6, WBLK = 6, KBLK = 32;
    assert(dy.ndim() == 4);
    // assert(dx.ndim() == 4);
    assert(ker.ndim() == 3);
    ssize_t N = dy.shape()[0], K = dy.shape()[1], H = dy.shape()[2], W = dy.shape()[3],
            C = ker.shape()[1];
    assert(ker.shape() == (shape_t{9, C, K}));
    Tensor dx = zeros({N, C, H, W});
    // assert(dx.shape() == (shape_t{N, C, H, W}));

    // BACK PASS: exchange C, K
    ssize_t nblkC = (K + CBLK - 1) / CBLK;
    ssize_t nblkH = (H + HBLK - 1) / HBLK;
    ssize_t nblkW = (W + WBLK - 1) / WBLK;
    ssize_t nblkK = (C + KBLK - 1) / KBLK;
    dim3 grid{unsigned(N * nblkC * nblkK), (unsigned)nblkH, (unsigned)nblkW};
    ssize_t block = 324;
    hipMemsetAsync(dx.data(), 0, sizeof(float) * dx.size());
    conv2d_3x3_dx_ker<CBLK, HBLK, WBLK, KBLK>
        <<<grid, block>>>(dy.data(), ker.data(), dx.data(), K, nblkC, H, nblkH,
                          W, nblkW, C, nblkK);
    return dx;
}

// we assume KBLK is a multiple of 4
template <ssize_t CBLK, ssize_t HBLK, ssize_t WBLK, ssize_t KBLK>
KERNEL conv2d_3x3_dk_ker(const float *x, const float *dy, float *dk, int C,
                      int nblkC, int H, int nblkH, int W, int nblkW, int K,
                      int nblkK) {
    __shared__ float t_x[CBLK][HBLK + 2][WBLK + 2], t_dy[KBLK][HBLK][WBLK], col[HBLK][WBLK][CBLK * 9];

    int batch = blockIdx.x;
    int off_k = (batch % nblkK) * KBLK;
    batch /= nblkK;
    int off_c = (batch % nblkC) * CBLK;
    batch /= nblkC;
    int off_h = blockIdx.y * HBLK, off_w = blockIdx.z * WBLK;
    int tid = threadIdx.x;
    x += batch * C * H * W;
    dy += batch * K * H * W;

    // load data
    // 256 threads
    if (tid < CBLK * (HBLK + 2) * (WBLK + 2)) {
        int c = tid / ((HBLK + 2) * (WBLK + 2));
        int h = (tid / (WBLK + 2)) % (HBLK + 2);
        int w = tid % (WBLK + 2);
        if (off_c + c < C && off_h + h < H && off_w + w < W)
            t_x[c][h][w] =
                x[(off_c + c) * H * W + (off_h + h) * W + off_w + w];
        else
            t_x[c][h][w] = 0.0;
    }

    // load gradient y
    // 288 threads
    if (tid < KBLK * HBLK * WBLK) {
        int k = tid / (HBLK * WBLK);
        int h = tid / WBLK % HBLK;
        int w = tid % WBLK;
        if (off_k + k < K && off_h + h < H && off_w + w < W)
            t_dy[k][h][w] = dy[(off_k + k) * H * W + (off_h + h) * W + off_w + w];
        else
            t_dy[k][h][w] = 0.0;
    }

    // im2col transformation
    // 324 threads
    if (tid < HBLK * WBLK * 9) {
        int phase = tid % 9, phase_h = phase / 3, phase_w = phase % 3;
        int _quot = tid / 9;
        int h = _quot / WBLK, w = _quot % WBLK;
        for (int c = 0; c < CBLK; ++c) {
            col[h][w][c * 9 + phase] = t_x[c][h + phase_h][w + phase_w];
        }
    }

    __syncthreads();

    // matrix multiplication
    // 288 threads
    if (tid < CBLK * KBLK * 9) {
        int k = tid % KBLK, c = (tid / KBLK) % CBLK, phase = tid / (KBLK * CBLK);
        float sum = 0;
        for (int h = 0; h < HBLK; ++h) {
            for (int w = 0; w < WBLK; ++w) {
                sum += col[h][w][c * 9 + phase] * t_dy[k][h][w];
            }
        }
        atomicAdd(&dk[phase * C * K + (off_c + c) * K + off_k + k], sum);
    }
}

Tensor conv2d_3x3_grad_k(const Tensor &dy, const Tensor &x) {
    assert(dy.ndim() == 4);
    assert(x.ndim() == 4);
    // assert(dk.ndim() == 3);
    ssize_t N = dy.shape()[0], K = dy.shape()[1], H = dy.shape()[2], W = dy.shape()[3],
            C = x.shape()[1];
    Tensor dk = zeros({9, C, K});
    // assert(dk.shape() == (shape_t{9, C, K}));
    assert(x.shape() == (shape_t{N, C, H, W}));
    constexpr int CBLK = 4, KBLK = 8, HBLK = 6, WBLK = 6;
    int nblkC = (C + CBLK - 1) / CBLK;
    int nblkH = (H + HBLK - 1) / HBLK;
    int nblkW = (W + WBLK - 1) / WBLK;
    int nblkK = (K + KBLK - 1) / KBLK;
    dim3 grid{unsigned(N * nblkC * nblkK), unsigned(nblkH), unsigned(nblkW)};
    ssize_t block = 324;
    hipMemsetAsync(dk.data(), 0, sizeof(float) * dk.size());
    conv2d_3x3_dk_ker<CBLK, HBLK, WBLK, KBLK>
        <<<grid, block>>>(x.data(), dy.data(), dk.data(), C, nblkC, H, nblkH,
                          W, nblkW, K, nblkK);
    return dk;
}
} // namespace ten
