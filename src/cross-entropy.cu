#include "hip/hip_runtime.h"
#include "macros.h"
#include "tensor.h"
#include <cassert>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <vector>

namespace ten {
void softmax(const Tensor &t, Tensor out) {
    assert(t.shape() == out.shape());
    assert(t.ndim() == 1 || t.ndim() == 2);
    assert(out.shape() == t.shape());
    ssize_t N = t.ndim() == 1 ? 1 : t.shape()[0], C = t.shape().back();

    for (ssize_t i = 0; i < N; ++i) {
        const float *L = t.data() + C * i, *R = t.data() + C * (i + 1);
        float mx = thrust::reduce(thrust::device, L, R, -INFINITY,
                                  thrust::maximum<float>());
        float exp_sum = thrust::transform_reduce(
            thrust::device, L, R,
            [=] __device__ __host__(float x) -> float {
                return std::exp(x - mx);
            },
            0.0, thrust::plus<float>());
        thrust::transform(thrust::device, L, R, out.data() + i * C,
                          [=] __device__ __host__(float x) {
                              return std::exp(x - mx) / exp_sum;
                          });
    }
}

float CELoss(const Tensor &t, std::vector<int> labels) {
    assert(t.ndim() == 2);
    ssize_t N = t.shape()[0], C = t.shape().back();

    Tensor sm = zeros(t.shape());
    softmax(t, sm);

    float sum = 0;
    for (ssize_t i = 0; i < N; ++i) {
        sum -= std::log(sm.at({i, labels[i]}));
    }
    return sum / N;
}

void CELoss_grad(const Tensor &t, std::vector<int> labels, Tensor dx) {
    assert(t.ndim() == 1 || t.ndim() == 2);
    assert(dx.shape() == dx.shape());
    ssize_t N = t.ndim() == 1 ? 1 : t.shape()[0], C = t.shape().back();

    softmax(t, dx);
    for (int i = 0; i < N; ++i) {
        dx.set({i, labels[i]}, dx.at({i, labels[i]}) - 1);
    }
    dx = dx * (1. / N);
}
} // namespace ten
